/************************************************************
Known issues:
This program only works on matrices smaller than or equal to 
256x256. 1024x1024 will cause segmentation faults and 512x512
simply causes the program to almost crash and return times of
0 for each kernel call. 
The matrices must be square and all matrices must be the same
size.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define MATSIZE 128
#define THREADS_PER_BLOCK 32 
 

 
//serial matrix multiplication kernel
__global__ void smultiply(int* g_a, int* g_b, int* g_c)
{
  int x, y, z;

  for (x = 0; x < MATSIZE; ++ x)
  {
    for (y = 0; y < MATSIZE; ++ y)
    {
      for (z = 0; z < MATSIZE; ++ z)
      {
	g_c[(x * MATSIZE) + y] += g_a[(x * MATSIZE) + z] * g_b[(z * MATSIZE) + y];
      }
    }
  }
}

//parallel matrix multiplication kernel
__global__ void pmultiply(double* g_a, double* g_b, double* g_d , int dim)
{
  int z;
  double sum = 0.0;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  for (z = 0; z < dim; ++ z)
  {
    sum += g_a[x * dim + z] * g_b[y + z * dim];
  }
    
  g_d[(x * dim) + y] = sum;
}

extern "C" void Cudamultiply(double* a, double* b, double* d, int Dim)
{
//  double *d;
  int i;
  double *g_a, *g_b, *g_d;
  int g_size = Dim * Dim * sizeof(double);
  hipEvent_t start, stop;
  float time; 

//  d=new double[Dim*Dim];

  hipEventCreate(&start);
  hipEventCreate(&stop);
  //used for timing the Cuda run

  hipMalloc(&g_a, g_size); //allocate memory on Cuda device
  hipMemcpy(g_a, a, g_size, hipMemcpyHostToDevice);
  //copy matrix A onto the Cuda device  

  hipMalloc(&g_b, g_size);
  hipMemcpy(g_b, b, g_size, hipMemcpyHostToDevice);
  
//  cudaMalloc(&g_c, g_size);
//  cudaMemcpy(g_c, c, g_size, cudaMemcpyHostToDevice);

  dim3 dimGrid((Dim / THREADS_PER_BLOCK), (Dim / THREADS_PER_BLOCK));
  //create the needed number of grids
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  //create the needed number of threads in each grid

  //serial Cuda kernel call
  //cudaEventRecord(start, 0);
  //smultiply<<<1,1>>>(g_a, g_b, g_c);
  //cudaEventRecord(stop, 0);  

  //cudaEventSynchronize(stop);
  //cudaEventElapsedTime(&time, start, stop);
  //get run time

//  cudaMemcpy(c, g_c, g_size, cudaMemcpyDeviceToHost);
  //copy results back to host device
//  cudaFree(g_c);  
  //free up unused user allocated memory on Cuda device

  printf("Time = %f milliseconds\n", time);
  
  //create a second answer matrix to use
  //This is not done until now so that memory on the
  //Cuda device is not wasted.
  hipMalloc(&g_d, g_size);
  hipMemcpy(g_d, d, g_size, hipMemcpyHostToDevice);

  //parallel Cuda kernel call
  hipEventRecord(start, 0);
  pmultiply<<<dimGrid,threads>>>(g_a, g_b, g_d,Dim);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("Time = %f milliseconds\n", time);

  hipMemcpy(d, g_d, g_size, hipMemcpyDeviceToHost);

  hipFree(g_a);
  hipFree(g_b);
  hipFree(g_d);
  //free up all unused user allocated memory on Cuda device

  //printf("\n");  

  /*The next 2 for loops print out the values of both
  answer matrices. This can be used to ensure that both
  kernel calls are producing the same results, and that
  the results are correct. This section can be commented 
  out when the user only wants the timing of a run.*/

//  for (i = 1; i <= Dim * Dim; ++ i)
//  {
//      if (i % Dim == 0)
//      {
////        printf("%f ", c[i-1]);
//	printf("\n");
//      }
//   
//      else
////        printf("%f ", c[i-1]);
//  }

 // printf("\n");

//  for (i = 1; i <= Dim * Dim; ++ i)
//  {
//    if ( i % Dim == 0)
//    {
//     printf("%f ", d[i-1]);
//     printf("\n");
//    }
//
//    else
//     printf("%f ", d[i-1]);
//  }
//  delete d;
}
