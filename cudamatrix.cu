#include "hip/hip_runtime.h"
/************************************************************
Known issues:
This program only works on matrices smaller than or equal to 
256x256. 1024x1024 will cause segmentation faults and 512x512
simply causes the program to almost crash and return times of
0 for each kernel call. 
The matrices must be square and all matrices must be the same
size.
The total number of threads needed (size of the matrix squared)
must be evenly divisible by the number of threads used per block. 
If a grid is declared that contains a number of threads less than
the declared number of threads per block, the kernel will return 
a value of 0 for the elements of the answer array corresponding
to all threads in that grid. For example, if the matrices being 
used are 20x20 and the declared number of threads per block is 
16x16, then the grids not containing 16x16 threads will return 0
for all elements. The kernel call will return a 20x20 matrix with
a 16x16 matrix inside it that is correct, and all the other 
elements will be 0.  
*/

#include <stdio.h>
#include <stdlib.h>
#define MATSIZE 128
#define THREADS_PER_BLOCK 32 
 

 
//serial matrix multiplication kernel
/****__global__ void smultiply(int* g_a, int* g_b, int* g_c)
{
  int x, y, z;

  for (x = 0; x < MATSIZE; ++ x)
  {
    for (y = 0; y < MATSIZE; ++ y)
    {
      for (z = 0; z < MATSIZE; ++ z)
      {
	g_c[(x * MATSIZE) + y] += g_a[(x * MATSIZE) + z] * g_b[(z * MATSIZE) + y];
      }
    }
  }
}*/

//parallel matrix multiplication kernel
__global__ void pmultiply(int* g_a, int* g_b, int* g_d , int dim)
{
  int z, sum = 0;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  for (z = 0; z < dim; ++ z)
  {
    sum += g_a[x * dim + z] * g_b[y + z * dim];
  }
    
  g_d[(x * dim) + y] = sum;
}

extern "C" void Cudamultiply(int* a, int* b, int* c, int Dim)
{
  
  int i, *g_a, *g_b, *g_c, *g_d;
  int g_size = Dim * Dim * sizeof(int);
  hipEvent_t start, stop;
  float time; 

 

  hipEventCreate(&start);
  hipEventCreate(&stop);
  //used for timing the Cuda run

  hipMalloc(&g_a, g_size); //allocate memory on Cuda device
  hipMemcpy(g_a, a, g_size, hipMemcpyHostToDevice);
  //copy matrix A onto the Cuda device  

  hipMalloc(&g_b, g_size);
  hipMemcpy(g_b, b, g_size, hipMemcpyHostToDevice);
  
  hipMalloc(&g_c, g_size);
  hipMemcpy(g_c, c, g_size, hipMemcpyHostToDevice);

  dim3 dimGrid((Dim / THREADS_PER_BLOCK), (Dim / THREADS_PER_BLOCK));
  //create the needed number of grids
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  //create the needed number of threads in each grid

  //serial Cuda kernel call
  //hipEventRecord(start, 0);
  //smultiply<<<1,1>>>(g_a, g_b, g_c);
  //hipEventRecord(stop, 0);  

  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  //get run time

  hipMemcpy(c, g_c, g_size, hipMemcpyDeviceToHost);
  //copy results back to host device
  hipFree(g_c);  
  //free up unused user allocated memory on Cuda device

  printf("Time = %f milliseconds\n", time);
  
  //create a second answer matrix to use
  //This is not done until now so that memory on the
  //Cuda device is not wasted.
  hipMalloc(&g_d, g_size);
  hipMemcpy(g_d, d, g_size, hipMemcpyHostToDevice);

  //parallel Cuda kernel call
  hipEventRecord(start, 0);
  pmultiply<<<dimGrid,threads>>>(g_a, g_b, g_d,Dim);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("Time = %f milliseconds\n", time);

  hipMemcpy(d, g_d, g_size, hipMemcpyDeviceToHost);

  hipFree(g_a);
  hipFree(g_b);
  hipFree(g_d);
  //free up all unused user allocated memory on Cuda device

  printf("\n");  

  /*The next 2 for loops print out the values of both
  answer matrices. This can be used to ensure that both
  kernel calls are producing the same results, and that
  the results are correct. This section can be commented 
  out when the user only wants the timing of a run.*/

  for (i = 1; i <= Dim * Dim; ++ i)
  {
      if (i % Dim == 0)
      {
        printf("%d ", c[i-1]);
	printf("\n");
      }
   
      else
        printf("%d ", c[i-1]);
  }

  printf("\n");

  for (i = 1; i <= Dim * Dim; ++ i)
  {
    if ( i % Dim == 0)
    {
     printf("%d ", d[i-1]);
     printf("\n");
    }

    else
     printf("%d ", d[i-1]);
  }
}